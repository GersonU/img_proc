#include "hip/hip_runtime.h"
#include "vector_add.hpp"

__global__ void vectorAddKernel(int* c, int SIZE) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < SIZE)
        c[i] = 3;
}

void vectorAdd(int* c, int SIZE) {
    vectorAddKernel<<<1, SIZE>>>(c, SIZE);
}