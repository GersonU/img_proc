#include "hip/hip_runtime.h"
#include "detect_objects.hpp"

using namespace cv;
using namespace cuda;

__global__ void cudaInRangeKernel(const PtrStepSz<uchar3> src, PtrStepSzb dst, int lbc0, int ubc0, int lbc1, int ubc1, int lbc2, int ubc2) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= src.cols || y >= src.rows) return;

	uchar3 pxl = src(y, x);

	if (pxl.x >= lbc0 && pxl.x <= ubc0 && pxl.y >= lbc1 && pxl.y <= ubc1 && pxl.z >= lbc2 && pxl.z <= ubc2) {
		dst(y, x) = 255;
	} else {
		dst(y, x) = 0;
	}
}

void cudaInRange(GpuMat &src, Scalar lwr_bnd, Scalar upr_bnd, GpuMat &dst) {
	const int m = 32;
	int rows = src.rows, cols = src.cols;
	if (rows == 0 || cols == 0) return;
	
	const dim3 grid(ceil((float) cols / m), ceil((float) rows / m), 1);
	const dim3 block(m, m, 1);

	cudaInRangeKernel<<<grid, block>>>(src, dst, lwr_bnd[0], upr_bnd[0], lwr_bnd[1], upr_bnd[1], lwr_bnd[2], upr_bnd[2]);
}